#include "hip/hip_runtime.h"
//one dimension FFT
#include "hip/hip_runtime.h"
#include ""
#include "Complex.cu"
#include <iostream>
#include <string>
#include <stdlib.h>
#include<stdio.h>
#include <time.h>


using namespace std;


int GetBits(int n) {
    int bits = 0;
    while (n >>= 1) {
        bits++;
    }
    return bits;
}

__device__ int br(int i, int bits) {
    int r = 0;
    do {
        r += i % 2 << --bits;
    } while (i /= 2);
    return r;
}

__device__ void Bufferfly(Complex* a, Complex* b, Complex factor) { 
    Complex a1 = (*a) + factor * (*b);
    Complex b1 = (*a) - factor * (*b);
    *a = a1;
    *b = b1;
}

__global__ void FFT(Complex nums[], Complex result[], int n, int bits) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= n) return;
    for (int i = 2; i < 2 * n; i *= 2) {
        if (tid % i == 0) {
            int k = i;
            if (n - tid < k) k = n - tid;
            for (int j = 0; j < k / 2; ++j) {
                Bufferfly(&nums[br(tid + j, bits)], &nums[br(tid + j + k / 2, bits)], Complex::W(k, j));
            }
        }
        __syncthreads();
    }
    result[tid] = nums[br(tid, bits)];
}

void printSequence(Complex nums[], const int N) {
    
    for (int i = 0; i < N; ++i) {
        double real = nums[i].real, imag = nums[i].imag;
        if (imag == 0) 
            printf("%.4f", real);
        else {
            if (imag > 0) 
                printf("%.4f+%.4fi", real, imag);
            else 
                printf("%.4f%.4fi", real, imag);
        }
        
        printf("\n");
    }
    
}

int main() {
    srand(time(0));

    const int TPB = 128;
    const int N = 128 * 32;        //FFT point
    
    const int bits = GetBits(N);

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    Complex* nums = (Complex*)malloc(sizeof(Complex) * N), * dNums, * dResult;
    for (int i = 0; i < N; ++i) {
        nums[i] = Complex::GetRandomReal();
    }
    printf("Length of Sequence: %d\n", N);
    cout<<"Before FFT"<<endl;
    printSequence(nums, N);

    hipMalloc((void**)&dNums, sizeof(Complex) * N);
    hipMalloc((void**)&dResult, sizeof(Complex) * N);
    hipMemcpy(dNums, nums, sizeof(Complex) * N, hipMemcpyHostToDevice);

    dim3 threadPerBlock(TPB);
    dim3 blockNum(N / TPB);
    

    //----------------------Start Record the time---------------------
    hipEventRecord(start, 0); //keep start time

    FFT << <threadPerBlock, blockNum >> > (dNums, dResult, N, bits);


    //----------------------END Record the time---------------------
    hipEventRecord(stop, 0); //keep stop time
    hipEventSynchronize(start); //wait stop event
    hipEventSynchronize(stop); //wait stop event
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time (GPU) : %f ms \n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //---------------------------------------------------------------
    hipMemcpy(nums, dResult, sizeof(Complex) * N, hipMemcpyDeviceToHost);

    cout<<"After FFT"<<endl;
    printSequence(nums, N);
    free(nums);
    hipFree(dNums);
    hipFree(dResult);

}