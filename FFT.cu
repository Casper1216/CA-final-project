#include "hip/hip_runtime.h"
//one dimension FFT
#include "hip/hip_runtime.h"
#include ""
#include "Complex.cu"
#include <iostream>
#include <string>
#include <stdlib.h>
#include<stdio.h>
#include <time.h>
#include<iostream>
#include<fstream>

using namespace std;
//#include <Windows.h>

int GetBits(int n) {
    int bits = 0;
    while (n >>= 1) {
        bits++;
    }
    return bits;
}

__device__ int br(int i, int bits) {
    int r = 0;
    do {
        r += i % 2 << --bits;
    } while (i /= 2);
    return r;
}

__device__ void Bufferfly(Complex *a, Complex *b, Complex factor) {
    Complex a1 = (*a) + factor * (*b);
    Complex b1 = (*a) - factor * (*b);
    *a = a1;
    *b = b1;
}

__global__ void FFT(Complex nums[], Complex result[], int n, int bits) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= n) return;
    for (int i = 2; i < 2 * n; i *= 2) {
        if (tid % i == 0) {
            int k = i;
            if (n - tid < k) k = n - tid;
            for (int j = 0; j < k / 2; ++j) {
                Bufferfly(&nums[br(tid + j, bits)], &nums[br(tid + j + k / 2, bits)], Complex::W(k, j));
            }
        }
        __syncthreads();
    }
    result[tid] = nums[br(tid, bits)];
}

void printSequence(Complex nums[], const int N) {
    printf("[");
    for (int i = 0; i < N; ++i) {
        double real = nums[i].real, imag = nums[i].imag;
        if (imag == 0) printf("%.16f", real);
        else {
            if (imag > 0) printf("%.16f+%.16fi", real, imag);
            else printf("%.16f%.16fi", real, imag);
        }
        if (i != N - 1) printf(", ");
    }
    printf("]\n");
}

int main() {
    srand(time(0));
    
    //const int TPB = 1024;
    //const int N = 1024 * 32;
    const int TPB = 128;
    const int N = 128 * 32;
    const int bits = GetBits(N);

    Complex *nums = (Complex*)malloc(sizeof(Complex) * N), *dNums, *dResult;
    for (int i = 0; i < N; ++i) {
        nums[i] = Complex::GetRandomReal();
    }
    printf("Length of Sequence: %d\n", N);
    // printf("Before FFT: \n");
    // printSequence(nums, N);
	

    // Start Record the time
    time_t  start = clock();
    //float s = GetTickCount();
    //***************************************************************************
    

    hipMalloc((void**)&dNums, sizeof(Complex) * N);
    hipMalloc((void**)&dResult, sizeof(Complex) * N);
    hipMemcpy(dNums, nums, sizeof(Complex) * N, hipMemcpyHostToDevice);
    
    dim3 threadPerBlock(TPB);
    dim3 blockNum((N + threadPerBlock.x - 1) / threadPerBlock.x);
    FFT<<<blockNum, threadPerBlock>>>(dNums, dResult, N, bits);

    hipMemcpy(nums, dResult, sizeof(Complex) * N, hipMemcpyDeviceToHost);

    //float cost = GetTickCount() - s;
    //printf("After FFT: \n");
    //printSequence(nums, N);
    //printf("Time of Transfromation: %fms", cost);
    // Record the end time
    time_t end = clock();
    double diff = end - start; // ms
    printf(" %f  sec\n", diff / CLOCKS_PER_SEC);

    printf("END \n");
    

//-------------write output-------------------------
	ofstream ofs;
	ofs.open("FFT_output.txt");
	if(!ofs.is_open()){
		cout<<"Fail to open"<<endl;
		return 1;	
	}
	ofs<<"[";
	for (int i = 0; i < N; ++i) {
		double real = nums[i].real, imag = nums[i].imag;
		if (imag == 0)
			ofs<<real;
		else {
		    if (imag > 0) 
			ofs<<real<<"+"<<imag<<"i";
		    else 
			ofs<<real<<imag<<"i";	//printf("%.16f%.16fi", real, imag);
		}
		if (i != N - 1) 
			ofs<<", ";
	}
	ofs<<"]\n";
//------------------------------------------------
    free(nums);
    hipFree(dNums);
    hipFree(dResult);
}
